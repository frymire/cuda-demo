/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "../common/book.h"

#define SIZE 64*1024*1024

float cuda_malloc_test(int size, bool up);
float cuda_host_alloc_test(int size, bool up);

int main(void) {

  float elapsedTime;
  const float MB = (float) 100*SIZE*sizeof(int)/1024/1024;
  
  // Try it with hipMalloc.
  elapsedTime = cuda_malloc_test(SIZE, true);
  printf("Copy up with hipMalloc: %3.1f ms, %3.1f MB/s\n", elapsedTime, MB/(elapsedTime/1000));

  elapsedTime = cuda_malloc_test(SIZE, false);
  printf("Copy down with hipMalloc: %3.1f ms, %3.1f MB/s\n", elapsedTime, MB/(elapsedTime/1000));

  // Now try it with hipHostAlloc to use page-locked memory on the CPU.
  elapsedTime = cuda_host_alloc_test(SIZE, true);
  printf("Copy up with cudaHostMalloc: %3.1f ms, %3.1f MB/s\n", elapsedTime, MB/(elapsedTime/1000));

  elapsedTime = cuda_host_alloc_test(SIZE, false);
  printf("Copy down with cudaHostMalloc: %3.1f ms, %3.1f MB/s\n", elapsedTime, MB/(elapsedTime/1000));
}

float cuda_malloc_test(int size, bool up) {

  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsedTime;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  a = (int*) malloc(size * sizeof(*a));
  HANDLE_NULL(a);
  HANDLE_ERROR(hipMalloc((void**) &dev_a, size * sizeof(*dev_a)));

  HANDLE_ERROR(hipEventRecord(start, 0));
  for (int i = 0; i<100; i++) {
    if (up)
      HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice));
    else
      HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost));
  }
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  free(a);
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  return elapsedTime;
}

float cuda_host_alloc_test(int size, bool up) {

  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsedTime;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  HANDLE_ERROR(hipHostAlloc((void**) &a, size*sizeof(*a), hipHostMallocDefault));
  HANDLE_ERROR(hipMalloc((void**) &dev_a, size * sizeof(*dev_a)));

  HANDLE_ERROR(hipEventRecord(start, 0));

  for (int i = 0; i<100; i++) {
    if (up)
      HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice));
    else
      HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(*a), hipMemcpyDeviceToHost));
  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  return elapsedTime;
}
