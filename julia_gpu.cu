#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 400

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex(float a, float b) : r(a), i(b) {}
	__device__ float magnitude2(void) { return r*r + i*i; }
	__device__ hipComplex operator*(const hipComplex& a) { return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i); }
	__device__ hipComplex operator+(const hipComplex& a) { return hipComplex(r + a.r, i + a.i); }
};

__device__ int julia(int x, int y) {

	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x) / (DIM/2);
	float jy = scale * (float)(DIM/2 - y) / (DIM/2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	// If the sequence diverges prior to 200 iterations, return 0.
	int i = 0;
	for (i = 0; i < 200; i++) {
		a = a*a + c;
		if (a.magnitude2() > 1000) return 0;
	}

	return 1;
}

__global__ void kernel(unsigned char *ptr) {

	// Map from blockIdx to pixel position.
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = gridDim.x * y + x;
	
	// Calculate the value at (x, y).
	int juliaValue = julia(x, y);

	// If the point is in the Julia set, set the corresponding pixel to red, otherwise black.
	ptr[offset*4 + 0] = 255*juliaValue;
	ptr[offset*4 + 1] = 0;
	ptr[offset*4 + 2] = 0;
	ptr[offset*4 + 3] = 255;
}

// Globals needed by the update routine
struct DataBlock {
	unsigned char *dev_bitmap;
};

int main(void) {

	DataBlock data;
	CPUBitmap bitmap(DIM, DIM, &data);
	unsigned char *dev_bitmap;

	printf("Image size = %d\n", bitmap.image_size());

	HANDLE_ERROR( hipMalloc((void**) &dev_bitmap, bitmap.image_size()) );
	data.dev_bitmap = dev_bitmap;

	// Split the work over a DIM x DIM grid, with 1 thread per block. (Probably very inefficient.)
	dim3 grid(DIM, DIM);
	kernel<<<grid, 1>>>(dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(dev_bitmap));

	bitmap.display_and_exit();	

	return 0;
}
