#include "hip/hip_runtime.h"
#include <stdlib.h>

#include "../common/book.h"

const int N = 32*1024;
const int nBytes = N*sizeof(int);

__global__ void vectorAdd(int *a, int *b, int *c);

int main(void) {

  // Allocate CPU memory.
  int *a, *b, *c;
  a = (int*) malloc(nBytes);
  b = (int*) malloc(nBytes);
  c = (int*) malloc(nBytes);

  // Fill arrays 'a' and 'b' on the CPU.
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = 2 * i;
  }

  // Allocate GPU memory.
  int *dev_a, *dev_b, *dev_c;
  HANDLE_ERROR(hipMalloc((void**) &dev_a, nBytes));
  HANDLE_ERROR(hipMalloc((void**) &dev_b, nBytes));
  HANDLE_ERROR(hipMalloc((void**) &dev_c, nBytes));

  // Copy the arrays to the GPU.
  HANDLE_ERROR(hipMemcpy(dev_a, a, nBytes, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, nBytes, hipMemcpyHostToDevice));

  vectorAdd<<<128, 1>>> (dev_a, dev_b, dev_c);

  // copy the array 'c' back from the GPU to the CPU
  HANDLE_ERROR(hipMemcpy(c, dev_c, nBytes, hipMemcpyDeviceToHost));

  // Display the first 5 results.
  for (int i = 0; i < 5; i++) {
  	printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  // Verify the result.
  bool success = true;
  for (int i = 0; i < N; i++) {
    if (a[i] + b[i] != c[i]) {
      printf("Error:  %d + %d != %d\n", a[i], b[i], c[i]);
      success = false;
    }
  }
  if (success) printf("The CPU and GPU vectors match.\n");

  // Free the GPU memory.
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  // Free the CPU memory.
  free(a);
  free(b);
  free(c);

  return 0;
}

__global__ void vectorAdd(int *a, int *b, int *c) {
  int threadID = blockIdx.x;
  while (threadID < N) {
    c[threadID] = a[threadID] + b[threadID];
    threadID += gridDim.x;
  }
}
