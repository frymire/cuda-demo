/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "../common/book.h"
#include "../common/cpu_bitmap.h"

const int dim = 1000;

struct hipComplex {
  float r;
  float i;
  hipComplex(float a, float b): r(a), i(b) {}
  float magnitude2(void) { return r*r + i*i; }
  hipComplex operator*(const hipComplex& a) { return hipComplex(r*a.r - i * a.i, i*a.r + r * a.i); }
  hipComplex operator+(const hipComplex& a) { return hipComplex(r + a.r, i + a.i); }
};


int julia(int x, int y);
void fillBitmapWithJuliaValues(unsigned char *bitmapData);

int main(void) {
  CPUBitmap bitmap(dim, dim);
  fillBitmapWithJuliaValues(bitmap.get_ptr());
  bitmap.display_and_exit();
}


void fillBitmapWithJuliaValues(unsigned char *bitmapData) {

  for (int y = 0; y < dim; y++) {
    for (int x = 0; x < dim; x++) {
      int offset = x + y*dim;
      bitmapData[offset*4 + 0] = 255*julia(x, y);
      bitmapData[offset*4 + 1] = 0;
      bitmapData[offset*4 + 2] = 0;
      bitmapData[offset*4 + 3] = 255;
    }
  }

}


int julia(int x, int y) {

  const float scale = 1.5;
  float jx = scale * (float) (dim/2 - x)/(dim/2);
  float jy = scale * (float) (dim/2 - y)/(dim/2);
  hipComplex a(jx, jy);

  hipComplex c(-0.8, 0.156);

  int i = 0;
  for (i = 0; i < 200; i++) {
    a = a*a + c;
    if (a.magnitude2() > 1000) return 0;
  }

  return 1;
}
