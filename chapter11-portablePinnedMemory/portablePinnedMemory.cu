#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

// NOTE: Compare this implementation to the multiGPU project, which does not use portable pinned host memory.

#include "../common/book.h"

#define imin(a, b) (a < b ? a : b)
#define N (33*1024*1024)
const int threadsPerBlock = 256;
const int nBlocks = imin(32, (N/2 + threadsPerBlock - 1) / threadsPerBlock);

struct DataStruct {
  int deviceID;
  int size;
  int offset;
  float* a;
  float* b;
  float returnValue;
};

unsigned WINAPI taskGPU(void *gpuTask);
__global__ void gpuComputeDotProduct(int size, float *a, float *b, float *c);

int main(void) {

  int deviceCount;
  HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
  if (deviceCount < 2) {
    printf("We need at least two compute 1.0 or greater devices, but only found %d\n", deviceCount);
    return 0;
  }

  hipDeviceProp_t gpuProperties;
  for (int i = 0; i < 2; i++) {
    HANDLE_ERROR(hipGetDeviceProperties(&gpuProperties, i));
    if (gpuProperties.canMapHostMemory != 1) {
      printf("Device %d can not map memory.\n", i);
      return 0;
    }
  }

  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

  // Pass the hipHostMallocPortable flag so that the host pointer can be used by multiple GPUs.
  float *a, *b;
  HANDLE_ERROR(hipHostAlloc((void**) &a, N*sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void**) &b, N*sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped));

  // Fill in the host memory with data.
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = 2*i;
  }

  // prepare for multithread
  DataStruct data[2];
  data[0].deviceID = 0;
  data[0].offset = 0;
  data[0].size = N/2;
  data[0].a = a;
  data[0].b = b;

  data[1].deviceID = 1;
  data[1].offset = N/2;
  data[1].size = N/2;
  data[1].a = a;
  data[1].b = b;

  CUTThread thread = start_thread(taskGPU, &(data[1]));
  taskGPU(&(data[0]));
  end_thread(thread);

  // Free CPU memory.
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));

  printf("Value calculated:  %f\n", data[0].returnValue + data[1].returnValue);
  printf("Should be:\t%f.\n", 27621693407370839851008.0f);
  return 0;
}

unsigned WINAPI taskGPU(void *gpuData) {

  DataStruct* data = (DataStruct*) gpuData;

  if (data->deviceID != 0) {
    HANDLE_ERROR(hipSetDevice(data->deviceID));
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
  }

  // Allocate CPU memory.
  float* a = data->a;
  float* b = data->b;
  float* partialC = (float*) malloc(nBlocks*sizeof(float));

  // Allocate GPU memory.
  float *gpuA, *gpuB, *gpuPartialC;
  HANDLE_ERROR(hipHostGetDevicePointer(&gpuA, a, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&gpuB, b, 0));
  HANDLE_ERROR(hipMalloc((void**) &gpuPartialC, nBlocks*sizeof(float)));

  // offset 'a' and 'b' to where this GPU is gets it data
  gpuA += data->offset;
  gpuB += data->offset;

  int size = data->size;
  gpuComputeDotProduct<<<nBlocks, threadsPerBlock>>>(size, gpuA, gpuB, gpuPartialC);

  // Copy array c from the GPU to the CPU.
  HANDLE_ERROR(hipMemcpy(partialC, gpuPartialC, nBlocks*sizeof(float), hipMemcpyDeviceToHost));

  // Complete the dot product calculation on the CPU.
  float dotProduct = 0;
  for (int i = 0; i < nBlocks; i++) { dotProduct += partialC[i]; }
  data->returnValue = dotProduct;

  HANDLE_ERROR(hipFree(gpuPartialC));
  free(partialC);

  return 0;
}

__global__ void gpuComputeDotProduct(int size, float *a, float *b, float *c) {

  __shared__ float cache[threadsPerBlock];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  float   temp = 0;
  while (tid < size) {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }

  // set the cache values
  cache[cacheIndex] = temp;

  // synchronize threads in this block
  __syncthreads();

  // for reductions, threadsPerBlock must be a power of 2
  // because of the following code
  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0)
    c[blockIdx.x] = cache[0];
}
