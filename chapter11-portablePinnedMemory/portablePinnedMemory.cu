#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

// NOTE: Compare this implementation to the multiGPU project, which does not use portable pinned host memory.

#include "../common/book.h"

#define imin(in0, in1) (in0 < in1 ? in0 : in1)
#define N (33*1024*1024)
const int threadsPerBlock = 256;
const int nBlocks = imin(32, (N/2 + threadsPerBlock - 1) / threadsPerBlock);

struct DataStruct {
  int deviceID;
  int size;
  int offset;
  float* in0;
  float* in1;
  float returnValue;
};

unsigned WINAPI taskGPU(void *gpuTask);
__global__ void gpuComputeDotProduct(int size, float *in0, float *in1, float *out);

int main(void) {

  int deviceCount;
  HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
  if (deviceCount < 2) {
    printf("We need at least two compute 1.0 or greater devices, but only found %d\n", deviceCount);
    return 0;
  }

  hipDeviceProp_t gpuProperties;
  for (int i = 0; i < 2; i++) {
    HANDLE_ERROR(hipGetDeviceProperties(&gpuProperties, i));
    if (!gpuProperties.canMapHostMemory) {
      printf("Device %d cannot map memory.\n", i);
      return 0;
    }
  }

  // It would be cleaner to do this at the task level within taskGPU(). However, we must first set 
  // the hipDeviceMapHost flag before we can allocate portable pinned memory using hipHostAlloc().
  HANDLE_ERROR(hipSetDevice(0));
  HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

  // Pass the hipHostMallocPortable flag so that the host pointer can be used by multiple GPUs. To use this
  // flag, you must have first called hipSetDevice().
  float *a, *b;
  HANDLE_ERROR(hipHostAlloc((void**) &a, N*sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void**) &b, N*sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped));

  // Fill in the host memory with data.
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = 2*i;
  }

  // Define task parameters for each GPU, passing CPU pointers to in0 and in1 on the GPU.

  DataStruct data[2];

  data[0].deviceID = 0;
  data[0].offset = 0;
  data[0].size = N/2;
  data[0].in0 = a;
  data[0].in1 = b;

  data[1].deviceID = 1;
  data[1].offset = N/2;
  data[1].size = N/2;
  data[1].in0 = a;
  data[1].in1 = b;

  CUTThread thread = start_thread(taskGPU, &(data[1]));
  taskGPU(&(data[0]));
  end_thread(thread);

  // Free CPU memory.
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));

  printf("Value calculated: \t%f\n", data[0].returnValue + data[1].returnValue);
  printf("Should be:\t\t%f.\n", 27621693407370839851008.0f);
  return 0;
}

unsigned WINAPI taskGPU(void *gpuData) {

  DataStruct* taskData = (DataStruct*) gpuData;

  // Again, it would be nicer if we could just set the device here, rather than in main. It was necessary
  // to set device 0 in main, however, so that we could allocate portable pinned host memory. Another subtle
  // point here though is that you can only call hipSetDevice() once per thread. Here, therefore, we have to
  // check whether we are already on device 0, since that was set in main.
  if (taskData->deviceID != 0) {
    HANDLE_ERROR(hipSetDevice(taskData->deviceID));
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
  }

  // Reference the data in CPU memory.
  float* a = taskData->in0;
  float* b = taskData->in1;
  float* partialC = (float*) malloc(nBlocks*sizeof(float));

  // Allocate GPU memory.
  float *gpuA, *gpuB, *gpuPartialC;
  HANDLE_ERROR(hipHostGetDevicePointer(&gpuA, a, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&gpuB, b, 0));
  HANDLE_ERROR(hipMalloc((void**) &gpuPartialC, nBlocks*sizeof(float)));

  // offset 'a' and 'b' to where this GPU is gets it data
  gpuA += taskData->offset;
  gpuB += taskData->offset;

  int size = taskData->size;
  gpuComputeDotProduct<<<nBlocks, threadsPerBlock>>>(size, gpuA, gpuB, gpuPartialC);

  // Copy array c from the GPU to the CPU.
  HANDLE_ERROR(hipMemcpy(partialC, gpuPartialC, nBlocks*sizeof(float), hipMemcpyDeviceToHost));

  // Complete the dot product calculation on the CPU.
  float dotProduct = 0;
  for (int i = 0; i < nBlocks; i++) { dotProduct += partialC[i]; }
  taskData->returnValue = dotProduct;

  HANDLE_ERROR(hipFree(gpuPartialC));
  free(partialC);

  return 0;
}

__global__ void gpuComputeDotProduct(int size, float *in0, float *in1, float *out) {

  __shared__ float cache[threadsPerBlock];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;

  float threadwiseDotProduct = 0;
  while (tid < size) {
    threadwiseDotProduct += in0[tid] * in1[tid];
    tid += blockDim.x * gridDim.x;
  }

  // set the cache values
  cache[cacheIndex] = threadwiseDotProduct;

  // synchronize threads in this block
  __syncthreads();

  // for reductions, threadsPerBlock must be a power of 2
  // because of the following code
  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIndex < i) { cache[cacheIndex] += cache[cacheIndex + i]; }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) { out[blockIdx.x] = cache[0]; }
}
