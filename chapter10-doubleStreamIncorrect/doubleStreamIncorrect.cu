#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "../common/book.h"

#define chunkSize 1024*1024
#define nBytesChunk chunkSize*sizeof(int)
#define nData 20*chunkSize
#define nBytesData nData*sizeof(int)
#define nThreadsPerBlock 256
#define nBlocks chunkSize/nThreadsPerBlock

__global__ void gpuComputeChunk(int* a, int* b, int* c);

int main(void) {

  hipDeviceProp_t gpuProperties;
  int whichDevice;
  HANDLE_ERROR(hipGetDevice(&whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties(&gpuProperties, whichDevice));
  if (!gpuProperties.deviceOverlap) {
    printf("Device will not handle overlaps, so no speed up from streams.\n");
    return 0;
  }

  hipEvent_t start, stop;
  float elapsedTime;

  hipStream_t stream0, stream1;
  int *cpuA, *cpuB, *cpuC;
  int *gpuA0, *gpuB0, *gpuC0;
  int *gpuA1, *gpuB1, *gpuC1;

  // Start the timers.
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  // Initialize the streams.
  HANDLE_ERROR(hipStreamCreate(&stream0));
  HANDLE_ERROR(hipStreamCreate(&stream1));

  // Allocate GPU memory.
  HANDLE_ERROR(hipMalloc((void**) &gpuA0, nBytesChunk));
  HANDLE_ERROR(hipMalloc((void**) &gpuB0, nBytesChunk));
  HANDLE_ERROR(hipMalloc((void**) &gpuC0, nBytesChunk));
  HANDLE_ERROR(hipMalloc((void**) &gpuA1, nBytesChunk));
  HANDLE_ERROR(hipMalloc((void**) &gpuB1, nBytesChunk));
  HANDLE_ERROR(hipMalloc((void**) &gpuC1, nBytesChunk));

  // Allocate host locked memory, used to stream.
  HANDLE_ERROR(hipHostAlloc((void**) &cpuA, nBytesData, hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**) &cpuB, nBytesData, hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**) &cpuC, nBytesData, hipHostMallocDefault));

  for (int i = 0; i < nData; i++) {
    cpuA[i] = rand();
    cpuB[i] = rand();
  }

  HANDLE_ERROR(hipEventRecord(start, 0));

  // Loop over the data in bite-sized chunks.
  for (int i = 0; i < nData; i += 2*chunkSize) {

    // Bad design here, done as a demo. Queuing all tasks for stream0 before all tasks for stream1.
    // (Actually, this gives the same performance on a 2080 Ti. Is the scheduler automatically rescheduling these?)

    // Queue all tasks for stream0  (bad move).
    HANDLE_ERROR(hipMemcpyAsync(gpuA0, cpuA + i, nBytesChunk, hipMemcpyHostToDevice, stream0));
    HANDLE_ERROR(hipMemcpyAsync(gpuB0, cpuB + i, nBytesChunk, hipMemcpyHostToDevice, stream0));
    gpuComputeChunk<<<nBlocks, nThreadsPerBlock, 0, stream0>>>(gpuA0, gpuB0, gpuC0);
    HANDLE_ERROR(hipMemcpyAsync(cpuC + i, gpuC0, nBytesChunk, hipMemcpyDeviceToHost, stream0));

    // Queue all tasks for stream1 (bad move).
    HANDLE_ERROR(hipMemcpyAsync(gpuA1, cpuA + i + chunkSize, nBytesChunk, hipMemcpyHostToDevice, stream1));
    HANDLE_ERROR(hipMemcpyAsync(gpuB1, cpuB + i + chunkSize, nBytesChunk, hipMemcpyHostToDevice, stream1));
    gpuComputeChunk<<<nBlocks, nThreadsPerBlock, 0, stream1>>>(gpuA1, gpuB1, gpuC1);
    HANDLE_ERROR(hipMemcpyAsync(cpuC + i + chunkSize, gpuC1, nBytesChunk, hipMemcpyDeviceToHost, stream1));
  }

  // Synch the streams to wait for the computations to finish.
  HANDLE_ERROR(hipStreamSynchronize(stream0));
  HANDLE_ERROR(hipStreamSynchronize(stream1));

  // Measure time.
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Time taken: %3.1f ms\n", elapsedTime);

  // Clean up the streams and memory.
  HANDLE_ERROR(hipHostFree(cpuA));
  HANDLE_ERROR(hipHostFree(cpuB));
  HANDLE_ERROR(hipHostFree(cpuC));
  HANDLE_ERROR(hipFree(gpuA0));
  HANDLE_ERROR(hipFree(gpuB0));
  HANDLE_ERROR(hipFree(gpuC0));
  HANDLE_ERROR(hipFree(gpuA1));
  HANDLE_ERROR(hipFree(gpuB1));
  HANDLE_ERROR(hipFree(gpuC1));
  HANDLE_ERROR(hipStreamDestroy(stream0));
  HANDLE_ERROR(hipStreamDestroy(stream1));

  return 0;
}

__global__ void gpuComputeChunk(int* a, int* b, int* c) {

  int i0 = blockIdx.x*blockDim.x + threadIdx.x;

  if (i0 < chunkSize) {
    int i1 = (i0 + 1) % nThreadsPerBlock;
    int i2 = (i0 + 2) % nThreadsPerBlock;
    c[i0] = (a[i0] + a[i1] + a[i2] + b[i0] + b[i1] + b[i2]) / 6.0f;
  }
}
