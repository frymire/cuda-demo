#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "../common/book.h"

#define imin(a, b) (a < b ? a : b)

const int dataLength = 33*1024*1024;
const int threadsPerBlock = 256;
const int nBlocks = imin(32, (dataLength + threadsPerBlock - 1) / threadsPerBlock);

__global__ void gpuComputeThreadwiseDotProduct(float *in0, float *in1, float *out);
float runDotProductTest(bool useZeroCopyMemory);

int main(void) {

  int gpuID;
  hipDeviceProp_t gpuProperties;
  HANDLE_ERROR(hipGetDevice(&gpuID));
  HANDLE_ERROR(hipGetDeviceProperties(&gpuProperties, gpuID));
  if (gpuProperties.canMapHostMemory != true) {
    printf("Device can not map memory.\n");
    return 0;
  }
  HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

  float elapsedTime;

  // Perform the test by using malloc to allocate memory on the CPU and transferring to the GPU as usual.
  elapsedTime = runDotProductTest(false);
  printf("Time using hipMalloc: %3.1f ms\n", elapsedTime);

  // Perform the test by using hipHostAlloc to allocate zero-copy memory on the CPU.
  elapsedTime = runDotProductTest(true);
  printf("Time using zero-copy memory with hipHostAlloc: %3.1f ms\n", elapsedTime);
}

float runDotProductTest(bool useZeroCopyMemory) {

  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  float *a, *b, *aTimesB, dotProduct;
  float *gpuA, *gpuB, *gpuATimesB;
  long nBytesData = dataLength*sizeof(float);

  if (useZeroCopyMemory) {

    // Allocate zero-copy memory on the CPU.
    HANDLE_ERROR(hipHostAlloc((void**) &a, nBytesData, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR(hipHostAlloc((void**) &b, nBytesData, hipHostMallocWriteCombined | hipHostMallocMapped));
    HANDLE_ERROR(hipHostAlloc((void**) &aTimesB, nBlocks*sizeof(float), hipHostMallocMapped));

    // Set the GPU pointers.
    HANDLE_ERROR(hipHostGetDevicePointer(&gpuA, a, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&gpuB, b, 0));
    HANDLE_ERROR(hipHostGetDevicePointer(&gpuATimesB, aTimesB, 0));

  } else {

    // Allocate CPU memory.
    a = (float*) malloc(nBytesData);
    b = (float*) malloc(nBytesData);
    aTimesB = (float*) malloc(nBlocks*sizeof(float));

    // Allocate GPU memory.
    HANDLE_ERROR(hipMalloc((void**) &gpuA, nBytesData));
    HANDLE_ERROR(hipMalloc((void**) &gpuB, nBytesData));
    HANDLE_ERROR(hipMalloc((void**) &gpuATimesB, nBlocks*sizeof(float)));
  }

  // Fill in the host memory with data.
  for (int i = 0; i < dataLength; i++) {
    a[i] = i;
    b[i] = i*2;
  }

  HANDLE_ERROR(hipEventRecord(start, 0));

  if (!useZeroCopyMemory) {
    // Copy arrays a and b to the GPU.
    HANDLE_ERROR(hipMemcpy(gpuA, a, nBytesData, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gpuB, b, nBytesData, hipMemcpyHostToDevice));
  }

  // Compute the dot product on the GPU.
  gpuComputeThreadwiseDotProduct<<<nBlocks, threadsPerBlock>>>(gpuA, gpuB, gpuATimesB);

  if (!useZeroCopyMemory) {
    // Copy array c back to the CPU.
    HANDLE_ERROR(hipMemcpy(aTimesB, gpuATimesB, nBlocks*sizeof(float), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipDeviceSynchronize());
  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  // Complete the sum on the CPU.
  dotProduct = 0;
  for (int i = 0; i < nBlocks; i++) { dotProduct += aTimesB[i]; }

  if (useZeroCopyMemory) {
    HANDLE_ERROR(hipHostFree(a)); // instead of hipFree, since the memory is on the CPU
    HANDLE_ERROR(hipHostFree(b));
    HANDLE_ERROR(hipHostFree(aTimesB));
  } else {
    HANDLE_ERROR(hipFree(gpuA));
    HANDLE_ERROR(hipFree(gpuB));
    HANDLE_ERROR(hipFree(gpuATimesB));
    free(a);
    free(b);
    free(aTimesB);
  }

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Value calculated: %f\n", dotProduct);
  return elapsedTime;
}

__global__ void gpuComputeThreadwiseDotProduct(float *in0, float *in1, float *out) {

  __shared__ float cache[threadsPerBlock];
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  int cacheIndex = threadIdx.x;

  float threadwiseDotProduct = 0;
  while (tid < dataLength) {
    threadwiseDotProduct += in0[tid]*in1[tid];
    tid += stride;
  }

  // Set the cache values and synchronize across threads before proceeding.
  cache[cacheIndex] = threadwiseDotProduct;
  __syncthreads();

  // for reductions, threadsPerBlock must be a power of 2 because of the following code
  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0) { out[blockIdx.x] = cache[0]; }
}
